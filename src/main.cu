#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <cmath>


#include "dbg.h"
#include "sphere.h"
#include "tracer.h"
#include "blobsys.h"





using namespace std;

__global__
void add(int n, float* x, float *y) {


    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}


int main() {

    //debug("Debug Mode ON!");


    config conf("/home/caesar/code/blinn/src/config.txt");

    tracer t(&conf);

    blobsys sys(&conf);


    scene s = sys.move(conf);
    for (int i = 0; i < conf.get_end_frame(); i++) {
        if (i >= conf.get_start_frame())
            t.meta_trace(s, i == (conf.get_end_frame() - 1));
		s = sys.move(conf);
    }


    return 0;
}
