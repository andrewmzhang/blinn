#include "hip/hip_runtime.h"
#include <thrust/tuple.h>
#include "cmath"
#include "dbg.h"
#include "tracer.h"
#include "render.h"
#include <memory>
#include <iostream>
#include <chrono>
#include "sphere.h"
using namespace std;

tracer::tracer(config *conf) {


    this->length = conf->get_width();
    this->squares.resize(length);

    double sq_width = 1.0 / length;
    for (int i = 0; i < length; i++) {
        this->squares[i].resize(length);
        for (int j = 0; j < length; j++) {
            this->squares[i][j].set(sq_width, i * sq_width, j * sq_width);
        }
    }


}
__device__
double meta_sec(sphere* spheres, sphere* b_spheres, int num_spheres, 
		ray *bullet, 
		uint32_t i, uint32_t j, 
		double &meta) {

    // Determine intersection
    double t_s = 1.0/0.0;
    double t_temp = t_s;

    double t1_s = -1.0/0.0;
    double t1_temp = t1_s;
    geometry *closest = nullptr;
    for (int s = 0; s < num_spheres; s++) {
        if (b_spheres[s].intersection(bullet, &t_temp, &t1_temp)) {
            if (t_temp < t_s)
                t_s = t_temp;
            if (t1_temp > t1_s)
                t1_s = t1_temp;
            closest = &spheres[s];
        }
    }
    if (closest == nullptr) {
        return -1;
    }

    double dist = t_s;
    while (dist < t1_s) {
        double meta_value = 0.0;
        for (int i = 0; i < num_spheres; i++)  {
			sphere sp = spheres[i];
			
			double x = bullet->pt->x + (dist * bullet->vec->x);
			double y = bullet->pt->y + (dist * bullet->vec->y);
			double z = bullet->pt->z + (dist * bullet->vec->z);

			double radius = sp.radius;

			double sp_meta = radius / 
				((x - sp.center.x) * (x - sp.center.x) + 
				 (y - sp.center.y) * (y - sp.center.y) + 
				 (z - sp.center.z) * (z - sp.center.z));
            
			meta_value += sp_meta;
        }

        meta = meta_value;
        if (meta_value > 10)
            return dist;

        dist += 0.0001;

    }
    return -1;

}

__device__
double meta_value(sphere& sp, double x, double y, double z) {

	double radius = sp.radius;

	double sp_meta = radius / 
		((x - sp.center.x) * (x - sp.center.x) + 
		 (y - sp.center.y) * (y - sp.center.y) + 
		 (z - sp.center.z) * (z - sp.center.z));
	return sp_meta;
}
__device__
thrust::tuple<double, double, double> approx_norm(double bx, double by, double bz, sphere* spheres, int num_spheres) {
    double d = 0.000000001;
    double tmp = 0.0;


    for (int s = 0; s < num_spheres; s++) {
        tmp += meta_value(spheres[s], bx + d, by, bz);
        tmp -= meta_value(spheres[s], bx, by, bz);

    }
    double normalx = tmp;

    tmp = 0.0;
    for (int s = 0; s < num_spheres; s++) {
        tmp += meta_value(spheres[s], bx, by + d,
                                           bz);
        tmp -= meta_value(spheres[s], bx, by, bz);
    }
    double normaly = (tmp);

    tmp = 0.0;
    for (int s = 0; s < num_spheres; s++) {
        tmp += meta_value(spheres[s], bx, by,
                                            bz + d);
        tmp -= meta_value(spheres[s], bx, by, bz);

    }
    double normalz = (tmp);

	double denom = sqrt(normalx * normalx + normaly*normaly + normalz*normalz);

	normalx /= denom;
	normaly /= denom;
	normalz /= denom;

    return thrust::make_tuple(normalx, normaly, normalz);
}



__global__
void gpu_meta_trace(
		sphere* spheres, int num_spheres,
		point light, double li,
		sphere* b_spheres, int num_b_spheres, 
		point* centers, render& r,
		int length) {

    for (int iter = 0; iter < length * length; iter++) {

        int i = iter / length;
        int j = iter % length;
		
        //color background(0);
		int backgroundR = 0;
		int backgroundG = 0;
		int backgroundB = 0;
        unique_ptr<point> camera = make_unique<point>(.5, .5, -5);
        point center = centers[iter];
        point vec = center - (*camera);
        unique_ptr<ray> bullet = make_unique<ray>(camera.get(), &vec);


        double meta;
        double dist = meta_sec(spheres, b_spheres, num_spheres, bullet.get(), i, j, meta);
		

        point bullet_loc = bullet->inch_by(dist);
        // Precondition: bullet_loc is equal to the intersection point
        if (dist > 0) {
			
			thrust::tuple<double, double, double> normal = approx_norm(bullet_loc.x, bullet_loc.y, bullet_loc.z, 
					spheres, num_spheres);// this->approx_norm(bullet_loc);
            double percent = 
				(thrust::get<0>(normal) * light.x) + 
				(thrust::get<1>(normal)  * light.y) + 
				(thrust::get<2>(normal)  * light.z);
            percent = (percent > 0) ? percent : 0.0;
            percent = 0.18 / M_PI * percent * li;

			double mixR = 0;
			double mixG = 0;
			double mixB = 0;

            for (int i = 0; i < num_spheres; i++) {
				sphere sp = spheres[i];
				double x = bullet_loc.x;
				double y = bullet_loc.y;
				double z = bullet_loc.z;

				double radius = sp.radius;

                double sp_meta = radius / 
					((x - sp.center.x) * (x - sp.center.x) + 
					 (y - sp.center.y) * (y - sp.center.y) + 
					 (z - sp.center.z) * (z - sp.center.z));
				double mult = sp_meta / meta;

				mixR += sp.c.R * mult;
				mixB += sp.c.B * mult;
				mixG += sp.c.G * mult;

            }

			backgroundR = mixR * percent;
			backgroundG = mixG * percent;
			backgroundB = mixB * percent;
        }

		int x = centers[iter].x;
		int y = centers[iter].y;
		r.frame[x][y][0] = backgroundR;
		r.frame[x][y][1] = backgroundG;
		r.frame[x][y][2] = backgroundB;
        
	}


}


void tracer::meta_trace(scene &scene) {

    this->count = scene.get_count();

    num_spheres = scene.get_spheres().size();
    spheres = scene.get_spheres();

    point light = scene.get_light();
    double li = scene.get_li();

    render r(this->length, this->length);

    log_info("Starting frame: %d", count);
    auto start = std::chrono::duration_cast<std::chrono::milliseconds>(
            std::chrono::system_clock::now().time_since_epoch()).count();

    this->b_spheres.clear();
    for (auto &sp : spheres) {
        b_spheres.push_back(sp.bound_radius(10.0 / num_spheres));
    }

	vector<square*> ptrs (squares.size());
	for (unsigned i=0, e=ptrs.size(); i<e; ++i) {
		    ptrs[i] = &(squares[i][0]); // assuming !thing[i].empty()
	}

	point* centers_flat = new point[length * length];

	for (int iter = 0; iter < length * length; iter++) {
		int i = iter / length;
		int j = iter % length;
		centers_flat[iter] = (&ptrs[0])[i][j].get_center();
	}

	sphere* gpu_spheres;
	sphere* gpu_b_spheres;
	point* gpu_centers;

	hipMallocManaged(&gpu_spheres, num_spheres * sizeof(sphere));
	hipMallocManaged(&gpu_b_spheres, num_spheres * sizeof(sphere));
	hipMallocManaged(&gpu_centers, (length * length) * sizeof(point));

	std::memcpy(gpu_spheres, spheres.data(), num_spheres * sizeof(sphere));
	std::memcpy(gpu_b_spheres, b_spheres.data(), num_spheres * sizeof(sphere));
	std::memcpy(gpu_centers, centers_flat, (length * length) * sizeof(point));
	
	gpu_meta_trace<<<1, 1>>>(
		gpu_spheres, num_spheres,
		light, li,
		gpu_b_spheres, b_spheres.size(), 
		gpu_centers, r, length);

	delete[] centers_flat;

// I want to move this to the gpu
// need to move r, b_sheres
/*
    for (int iter = 0; iter < length * length; iter++) {

        int i = iter / length;
        int j = iter % length;

        color background(125, 125, 125);
        unique_ptr<point> camera = make_unique<point>(.5, .5, -5);
        color shade = background;
        point center = this->squares[i][j].get_center();
        point vec = center - (*camera);
        unique_ptr<ray> bullet = make_unique<ray>(camera.get(), &vec);


        double meta;
        double dist = this->meta_sec(bullet.get(), i, j, meta);
        point bullet_loc = bullet->inch_by(dist);

        // Precondition: bullet_loc is equal to the intersection point
        if (dist > 0) {

            point normal = this->approx_norm(bullet_loc);
            double percent = (normal * light);
            percent = std::max(0.0, percent);
            percent = 0.18 / M_PI * percent * li;

            color mix;

            for (auto sp : spheres) {
                double sp_meta = sp.meta_value(bullet_loc);
                mix = mix + sp.get_color() * (sp_meta / meta);
            }


            shade = mix * percent;

        }

        square *s = &this->squares[i][j];
        r.set_point(s->get_center().get_x(), s->get_center().get_y(), &shade);

    }
	*/

	// End of tracing code

    auto end = std::chrono::duration_cast<std::chrono::milliseconds>(
            std::chrono::system_clock::now().time_since_epoch()).count();
    double time = ((double) (end - start)) / 1000.0;
    fflush(stderr);
    log_info("Time elapsed: %.3fs, %.3f px/sec", time, length * length / time);
    fflush(stderr);

    r.print(count, "hello.bmp");

    count++;
}


double tracer::meta_sec(ray *bullet, uint32_t i, uint32_t j, double &meta) {

    // Determine intersection
    double t_s = numeric_limits<double>::infinity();
    double t_temp = t_s;

    double t1_s = -numeric_limits<double>::infinity();
    double t1_temp = t1_s;
    geometry *closest = nullptr;
    for (int s = 0; s < num_spheres; s++) {
        if (this->b_spheres[s].intersection(bullet, &t_temp, &t1_temp)) {
            if (t_temp < t_s)
                t_s = t_temp;
            if (t1_temp > t1_s)
                t1_s = t1_temp;
            closest = &this->spheres[s];
        }
    }
    if (closest == nullptr) {
        return -1;
    }

    double dist = t_s;
    while (dist < t1_s) {
        double meta_value = 0.0;
        for (geometry &s : this->spheres) {
            meta_value += s.meta_value(bullet->inch_by(dist));
        }

        meta = meta_value;
        if (meta_value > 10)
            return dist;

        dist += 0.0001;

    }
    return -1;

}

point tracer::approx_norm(point bullet_loc) {
    point normal(0, 0, 0);
    double d = 0.000000001;
    double tmp = 0.0;

    for (int s = 0; s < num_spheres; s++) {
        tmp += this->spheres[s].meta_value(bullet_loc.get_x() + d, bullet_loc.get_y(),
                                           bullet_loc.get_z());
        tmp -= this->spheres[s].meta_value(bullet_loc.get_x(), bullet_loc.get_y(), bullet_loc.get_z());

    }
    normal.set_x(tmp);

    tmp = 0.0;
    for (int s = 0; s < num_spheres; s++) {
        tmp += this->spheres[s].meta_value(bullet_loc.get_x(), bullet_loc.get_y() + d,
                                           bullet_loc.get_z());
        tmp -= this->spheres[s].meta_value(bullet_loc.get_x(), bullet_loc.get_y(), bullet_loc.get_z());
    }
    normal.set_y(tmp);

    tmp = 0.0;
    for (int s = 0; s < num_spheres; s++) {
        tmp += this->spheres[s].meta_value(bullet_loc.get_x(), bullet_loc.get_y(),
                                            bullet_loc.get_z() + d);
        tmp -= this->spheres[s].meta_value(bullet_loc.get_x(), bullet_loc.get_y(), bullet_loc.get_z());

    }
    normal.set_z(tmp);
    normal.normalize();
    return normal;
}


void tracer::trace() {

    render r(this->length, this->length);
    color background(125, 125, 125);

    int length = this->length;

    unique_ptr<point> camera = make_unique<point>(.5, .5, -5);


    for (int i = 0; i < length; i++) {
        for (int j = 0; j < length; j++) {

            color shade = background;
            point center = this->squares[i][j].get_center();
            point vec = center - (*camera);
            unique_ptr<ray> bullet = make_unique<ray>(camera.get(), &vec);


            double t_s = numeric_limits<double>::infinity();
            double t_temp = t_s;
            geometry *closest = nullptr;
            for (int s = 0; s < num_spheres; s++) {
                if (this->spheres[s].intersection(bullet.get(), &t_temp) && t_temp < t_s) {
                    t_s = t_temp;
                    closest = &this->spheres[s];
                }
            }

            if (closest != nullptr) {

                point surface = *(bullet->pt) + *(bullet->vec) * t_s;

                point normal = closest->get_normal(&surface);

                double percent = -(normal * light);
                percent = std::max(0.0, percent);
                percent = closest->albedo / M_PI * percent * this->li;
                //debug("Light percentage %f", percent);

                shade = closest->get_color() * percent;
            }


            square *s = &this->squares[i][j];
            this->squares[i][j].set_value(&shade);
            r.set_color(&shade);
            r.set_point(s->get_center().get_x(), s->get_center().get_y());


        }
    }

    r.print(count, "hello.bmp");

    count++;
}
