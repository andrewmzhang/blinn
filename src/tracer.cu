#include "hip/hip_runtime.h"
#include <thrust/tuple.h>
#include "cmath"
#include "dbg.h"
#include "tracer.h"
#include "render.h"
#include <memory>
#include <iostream>
#include <chrono>
#include "sphere.h"
using namespace std;

tracer::tracer(config *conf) {


    this->length = conf->get_width();
    this->squares.resize(length);

    double sq_width = 1.0 / length;
    for (int i = 0; i < length; i++) {
        this->squares[i].resize(length);
        for (int j = 0; j < length; j++) {
            this->squares[i][j].set(sq_width, i * sq_width, j * sq_width);
        }
    }


}

struct gcolor {
	int R;
	int G;
	int B;
};

struct gsphere {

	double radius;
	double k;

	point center;
	gcolor c;

	bool meta;
};

struct gpoint {

	double x;
	double y;
	double z;

};

typedef struct gray {

	gpoint pt;
	gpoint vec;

} gray;

__device__
double dot_product(const gpoint& a, const gpoint& b) {

	return a.x*b.x + a.y*b.y + a.z*b.z;
}

__device__
gpoint normalize(gpoint& a) {
	double denom = sqrt(dot_product(a, a));
	return {a.x / denom, a.y/denom, a.z/denom};
}



__device__
bool intersection(gsphere& s, gray& r, double* t, double* u) {

	gpoint L = {s.center.x - r.pt.x, s.center.y - r.pt.y, s.center.z - r.pt.z};

	double ldotvec = dot_product(L, normalize(r.vec));
	
	double d2 = dot_product(L, L) - (ldotvec * ldotvec);
	double radius2 = s.radius * s.radius;

	if (d2 > radius2) return false;
	double thc = sqrt(radius2 - d2);

	double t0 = ldotvec - thc;
	double t1 = ldotvec + thc;

	if (t0 > t1) {
		double temp = t0;
		t0 = t1;
		t1 = temp;
	}

	if (t0 < 0) {
		t0 = t1;
		if (t0 < 0)
			return false;
	}

	*t = t0;
	*u = t1;

	return true;

}




__device__
double meta_sec(gsphere* spheres, gsphere* b_spheres, int num_spheres, 
		gray bullet, 
		int i, int j, 
		double &meta) {

    // Determine intersection
    double t_s = 9999999999;
    double t_temp = t_s;

    double t1_s = -9999999;
    double t1_temp = t1_s;
    gsphere *closest = nullptr;
    for (int s = 0; s < num_spheres; s++) {
        if (intersection(b_spheres[s], bullet, &t_temp, &t1_temp)) {
            if (t_temp < t_s)
                t_s = t_temp;
            if (t1_temp > t1_s)
                t1_s = t1_temp;
            closest = &spheres[s];
        }
    }
    if (closest == nullptr) {
        return -1;
    }

    double dist = t_s;
    while (dist < t1_s) {
        double meta_value = 0.0;
        for (int i = 0; i < num_spheres; i++)  {
			gsphere& sp = spheres[i];
			
			double x = bullet.pt.x + (dist * bullet.vec.x);
			double y = bullet.pt.y + (dist * bullet.vec.y);
			double z = bullet.pt.z + (dist * bullet.vec.z);

			double radius = sp.radius;

			double sp_meta = radius / 
				((x - sp.center.x) * (x - sp.center.x) + 
				 (y - sp.center.y) * (y - sp.center.y) + 
				 (z - sp.center.z) * (z - sp.center.z));
            
			meta_value += sp_meta;
        }

        meta = meta_value;
        if (meta_value > 10)
            return dist;

        dist += 0.0001;

    }
    return -1;

}

__device__
double meta_value(gsphere& sp, double x, double y, double z) {

	double radius = sp.radius;

	double sp_meta = radius / 
		((x - sp.center.x) * (x - sp.center.x) + 
		 (y - sp.center.y) * (y - sp.center.y) + 
		 (z - sp.center.z) * (z - sp.center.z));
	return sp_meta;
}
__device__
thrust::tuple<double, double, double> approx_norm(double bx, double by, double bz, gsphere* spheres, int num_spheres) {
    double d = 0.000000001;
    double tmp = 0.0;


    for (int s = 0; s < num_spheres; s++) {
        tmp += meta_value(spheres[s], bx + d, by, bz);
        tmp -= meta_value(spheres[s], bx, by, bz);

    }
    double normalx = tmp;

    tmp = 0.0;
    for (int s = 0; s < num_spheres; s++) {
        tmp += meta_value(spheres[s], bx, by + d,
                                           bz);
        tmp -= meta_value(spheres[s], bx, by, bz);
    }
    double normaly = (tmp);

    tmp = 0.0;
    for (int s = 0; s < num_spheres; s++) {
        tmp += meta_value(spheres[s], bx, by,
                                            bz + d);
        tmp -= meta_value(spheres[s], bx, by, bz);

    }
    double normalz = (tmp);

	double denom = sqrt(normalx * normalx + normaly*normaly + normalz*normalz);

	normalx /= denom;
	normaly /= denom;
	normalz /= denom;

    return thrust::make_tuple(normalx, normaly, normalz);
}


__device__
gpoint inch_by(gray b, double x) {
	
	double startx = b.pt.x;
	double starty = b.pt.y;
	double startz = b.pt.z;

	double deltax = b.vec.x;
	double deltay = b.vec.y;
	double deltaz = b.vec.z;


	return {startx + x * deltax, starty + x * deltay, startz + x * deltaz};
}


__global__
void gpu_meta_trace(
		gsphere* spheres, int num_spheres,
		
		gpoint light, double li,
		
		gsphere* b_spheres, int num_b_spheres, 
		
		gpoint* centers,
		int length,
		int* r) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;


    for (int iter = index; iter < length * length; iter += stride) {

        int i = iter / length;
        int j = iter % length;
		
        //color background(0);
		int backgroundR = 125;
		int backgroundG = 125;
		int backgroundB = 125;
        gpoint camera = {.5, .5, -5};
        gpoint center = centers[iter];
		gpoint vec = {center.x - camera.x, center.y - camera.y, center.z - camera.z};



        double meta;
		gpoint nvec = normalize(vec);
		gray gbullet = {camera, nvec};
        double dist = meta_sec(spheres, b_spheres, num_spheres, gbullet, i, j, meta);
		

        gpoint bullet_loc = inch_by(gbullet, dist);
        // Precondition: bullet_loc is equal to the intersection point
        if (dist > 0) {
			
			thrust::tuple<double, double, double> normal = approx_norm(bullet_loc.x, bullet_loc.y, bullet_loc.z, 
					spheres, num_spheres);// this->approx_norm(bullet_loc);
            double percent = 
				(thrust::get<0>(normal) * light.x) + 
				(thrust::get<1>(normal)  * light.y) + 
				(thrust::get<2>(normal)  * light.z);
            percent = (percent > 0) ? percent : 0.0;
            percent = 0.18 / M_PI * percent * li;

			double mixR = 0;
			double mixG = 0;
			double mixB = 0;

            for (int i = 0; i < num_spheres; i++) {
				gsphere& sp = spheres[i];
				double x = bullet_loc.x;
				double y = bullet_loc.y;
				double z = bullet_loc.z;

				double radius = sp.radius;

                double sp_meta = radius / 
					((x - sp.center.x) * (x - sp.center.x) + 
					 (y - sp.center.y) * (y - sp.center.y) + 
					 (z - sp.center.z) * (z - sp.center.z));
				double mult = sp_meta / meta;

				mixR += sp.c.R * mult;
				mixB += sp.c.B * mult;
				mixG += sp.c.G * mult;

            }

			backgroundR = mixR * percent;
			backgroundG = mixG * percent;
			backgroundB = mixB * percent;
        }

		double x = centers[iter].x;
		double y = centers[iter].y;


		int point_x = (int) round(x * (length - 1));
		int point_y = (int) round(y * (length - 1));

		int set = point_x*length*3 + point_y*3;

		r[set + 0] = backgroundR;
		r[set + 1] = backgroundG;
		r[set + 2] = backgroundB;


		//r.frame[x][y][0] = backgroundR;
		//r.frame[x][y][1] = backgroundG;
		//r.frame[x][y][2] = backgroundB;
        
	}


}


void tracer::meta_trace(scene &scene, bool clean) {

    this->count = scene.get_count();

    num_spheres = scene.get_spheres().size();
    spheres = scene.get_spheres();

    point n_light = scene.get_light();
	gpoint light = {n_light.x, n_light.y, n_light.z};
    double li = scene.get_li();

    render r(this->length, this->length);

    log_info("Starting frame: %d", count);
    auto start = std::chrono::duration_cast<std::chrono::milliseconds>(
            std::chrono::system_clock::now().time_since_epoch()).count();

    this->b_spheres.clear();
    for (auto &sp : spheres) {
        b_spheres.push_back(sp.bound_radius(10.0 / num_spheres));
    }

	vector<square*> ptrs (squares.size());
	for (unsigned i=0, e=ptrs.size(); i<e; ++i) {
		    ptrs[i] = &(squares[i][0]); // assuming !thing[i].empty()
	}

	point* centers_flat = new point[length * length];

	for (int iter = 0; iter < length * length; iter++) {
		int i = iter / length;
		int j = iter % length;
		centers_flat[iter] = (&ptrs[0])[i][j].get_center();
	}

	static gsphere* gpu_spheres;
	static gsphere* gpu_b_spheres;
	static gpoint* gpu_centers;
	static int* the_frame;
	static bool first = 0;

	if (!first) {
		hipMallocManaged(&gpu_spheres, num_spheres*sizeof(gsphere));
		hipMallocManaged(&gpu_b_spheres, num_spheres * sizeof(gsphere));
		hipMallocManaged(&gpu_centers, (length * length) * sizeof(gpoint));
		hipMallocManaged(&the_frame, (length * length * 3) * sizeof(int));
		first = true;
	}
		
	for (int i = 0; i < num_spheres; i++) {
		sphere s = spheres.at(i);
		gpu_spheres[i].radius = s.radius;
		gpu_spheres[i].k = s.k;
		gpu_spheres[i].center = s.center;
		gpu_spheres[i].c = {s.c.R, s.c.G, s.c.B};
		gpu_spheres[i].meta = s.meta;
	}
	
	for (int i = 0; i < num_spheres; i++) {
		sphere s = b_spheres.at(i);
		gpu_b_spheres[i].radius = s.radius;
		gpu_b_spheres[i].k = s.k;
		gpu_b_spheres[i].center = s.center;
		gpu_b_spheres[i].c= {s.c.R, s.c.G, s.c.B};
		gpu_b_spheres[i].meta = s.meta;
	}
	
	for (int i = 0; i < length * length; i++) {
		point& p = centers_flat[i];
		gpu_centers[i] = {p.x, p.y, p.z};
	}

	delete[] centers_flat;
	int numBlocks = ((length*length) + 256 - 1) / 256;
	gpu_meta_trace<<<numBlocks, 256>>>(
		gpu_spheres, num_spheres,
		light, li,
		gpu_b_spheres, num_spheres, 
		gpu_centers, length, the_frame);

	hipDeviceSynchronize();
    auto end = std::chrono::duration_cast<std::chrono::milliseconds>(
            std::chrono::system_clock::now().time_since_epoch()).count();
    double time = ((double) (end - start)) / 1000.0;
    log_info("Time elapsed: %.3fs, %.3f px/sec", time, length * length / time);

	r.set_frame(the_frame, length);
	r.print(count, "hello.bmp");

	if (clean) {	
		hipFree(gpu_spheres);
		hipFree(gpu_b_spheres);
		hipFree(gpu_centers);
		hipFree(the_frame);
	}
	log_info("Finished frame");
    count++;
	log_info("%d", count);
}


double tracer::meta_sec(ray *bullet, uint32_t i, uint32_t j, double &meta) {

    // Determine intersection
    double t_s = numeric_limits<double>::infinity();
    double t_temp = t_s;

    double t1_s = -numeric_limits<double>::infinity();
    double t1_temp = t1_s;
    geometry *closest = nullptr;
    for (int s = 0; s < num_spheres; s++) {
        if (this->b_spheres[s].intersection(bullet, &t_temp, &t1_temp)) {
            if (t_temp < t_s)
                t_s = t_temp;
            if (t1_temp > t1_s)
                t1_s = t1_temp;
            closest = &this->spheres[s];
        }
    }
    if (closest == nullptr) {
        return -1;
    }

    double dist = t_s;
    while (dist < t1_s) {
        double meta_value = 0.0;
        for (geometry &s : this->spheres) {
            meta_value += s.meta_value(bullet->inch_by(dist));
        }

        meta = meta_value;
        if (meta_value > 10)
            return dist;

        dist += 0.0001;

    }
    return -1;

}

point tracer::approx_norm(point bullet_loc) {
    point normal(0, 0, 0);
    double d = 0.000000001;
    double tmp = 0.0;

    for (int s = 0; s < num_spheres; s++) {
        tmp += this->spheres[s].meta_value(bullet_loc.get_x() + d, bullet_loc.get_y(),
                                           bullet_loc.get_z());
        tmp -= this->spheres[s].meta_value(bullet_loc.get_x(), bullet_loc.get_y(), bullet_loc.get_z());

    }
    normal.set_x(tmp);

    tmp = 0.0;
    for (int s = 0; s < num_spheres; s++) {
        tmp += this->spheres[s].meta_value(bullet_loc.get_x(), bullet_loc.get_y() + d,
                                           bullet_loc.get_z());
        tmp -= this->spheres[s].meta_value(bullet_loc.get_x(), bullet_loc.get_y(), bullet_loc.get_z());
    }
    normal.set_y(tmp);

    tmp = 0.0;
    for (int s = 0; s < num_spheres; s++) {
        tmp += this->spheres[s].meta_value(bullet_loc.get_x(), bullet_loc.get_y(),
                                            bullet_loc.get_z() + d);
        tmp -= this->spheres[s].meta_value(bullet_loc.get_x(), bullet_loc.get_y(), bullet_loc.get_z());

    }
    normal.set_z(tmp);
    normal.normalize();
    return normal;
}


void tracer::trace() {

    render r(this->length, this->length);
    color background(125, 125, 125);

    int length = this->length;

    unique_ptr<point> camera = make_unique<point>(.5, .5, -5);


    for (int i = 0; i < length; i++) {
        for (int j = 0; j < length; j++) {

            color shade = background;
            point center = this->squares[i][j].get_center();
            point vec = center - (*camera);
            unique_ptr<ray> bullet = make_unique<ray>(camera.get(), &vec);


            double t_s = numeric_limits<double>::infinity();
            double t_temp = t_s;
            geometry *closest = nullptr;
            for (int s = 0; s < num_spheres; s++) {
                if (this->spheres[s].intersection(bullet.get(), &t_temp) && t_temp < t_s) {
                    t_s = t_temp;
                    closest = &this->spheres[s];
                }
            }

            if (closest != nullptr) {

                point surface = *(bullet->pt) + *(bullet->vec) * t_s;

                point normal = closest->get_normal(&surface);

                double percent = -(normal * light);
                percent = std::max(0.0, percent);
                percent = closest->albedo / M_PI * percent * this->li;
                //debug("Light percentage %f", percent);

                shade = closest->get_color() * percent;
            }


            square *s = &this->squares[i][j];
            this->squares[i][j].set_value(&shade);
            r.set_color(&shade);
            r.set_point(s->get_center().get_x(), s->get_center().get_y());


        }
    }

    r.print(count, "hello.bmp");

    count++;
}

// I want to move this to the gpu
// need to move r, b_sheres
/*
    for (int iter = 0; iter < length * length; iter++) {

        int i = iter / length;
        int j = iter % length;

        color background(125, 125, 125);
        unique_ptr<point> camera = make_unique<point>(.5, .5, -5);
        color shade = background;
        point center = this->squares[i][j].get_center();
        point vec = center - (*camera);
        unique_ptr<ray> bullet = make_unique<ray>(camera.get(), &vec);


        double meta;
        double dist = this->meta_sec(bullet.get(), i, j, meta);
        point bullet_loc = bullet->inch_by(dist);

        // Precondition: bullet_loc is equal to the intersection point
        if (dist > 0) {

            point normal = this->approx_norm(bullet_loc);
            double percent = (normal * light);
            percent = std::max(0.0, percent);
            percent = 0.18 / M_PI * percent * li;

            color mix;

            for (auto sp : spheres) {
                double sp_meta = sp.meta_value(bullet_loc);
                mix = mix + sp.get_color() * (sp_meta / meta);
            }


            shade = mix * percent;

        }

        square *s = &this->squares[i][j];
        r.set_point(s->get_center().get_x(), s->get_center().get_y(), &shade);

    }
	*/

	// End of tracing code
